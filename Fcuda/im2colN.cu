
#include <cmath>
#include <cstdlib>
#include <cstring>
#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void im2colN(float* data_col, const float* ori_data,
  const int height, const int width, const int channels, const int ksize,
  const int height_col, const int width_col) {

  int poolx = threadIdx.x + blockIdx.x * blockDim.x;
  int pooly = threadIdx.y + blockIdx.y * blockDim.y;
  int poolz = blockIdx.z;
  if (pooly < height_col && poolx < width_col) {

    int hstart = pooly;
    int hend = pooly + ksize;
    int wstart = poolx;
    int wend = poolx + ksize;
    int patchNum = pooly * height_col + poolx;
    ori_data += poolz * height * width;
    data_col += height_col * width_col * poolz;
    data_col += patchNum;
 
    for (int h = hstart; h < hend; h++) {
      for (int w = wstart; w < wend; w++) {
             
             *data_col = ori_data[h * width + w];
              data_col += height_col * width_col * channels;

         }
      }
    }
}


