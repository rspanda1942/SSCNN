// Copyright 2013 Yangqing Jia

#include <cmath>
#include <cstdlib>
#include <cstring>
#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void im2col(const int n, const float * data_im,
  const int height, const int width, const int ksize,
  const int stride, const int height_col, const int width_col, float * data_col) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    int w_out = index % width_col;
    index /= width_col;
    int h_out = index % height_col;
    int channel_in = index / height_col;
    int channel_out = channel_in * ksize * ksize;
    int h_in = h_out * stride;
    int w_in = w_out * stride;
    data_col += (channel_out * height_col + h_out) * width_col + w_out;
    data_im += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize; ++i) {
      for (int j = 0; j < ksize; ++j) {
        *data_col = data_im[i * width + j];
        data_col += height_col * width_col;
      }
    }
  }
}



