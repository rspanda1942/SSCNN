

#include <cmath>
#include <cstdlib>
#include <cstring>
#include "hip/hip_runtime.h"


#include <stdio.h>

// -------------------------------------------------------------------------------------
__global__ void FMaxPoolForward(const float* ori_data, float* pool_data, int* indice_data,
    const int num, const int channels, 
    const int height, const int width, 
    const int pooled_height, const int pooled_width,
    const int poolsize, const int poolstrike) {

  int poolx = threadIdx.x + blockIdx.x * blockDim.x;
  int pooly = threadIdx.y + blockIdx.y * blockDim.y;
  int poolz = blockIdx.z;

  if (pooly < pooled_height && poolx < pooled_width) {

    int hstart = pooly * poolstrike;
    int hend = hstart + poolsize;
    int wstart = poolx * poolstrike;
    int wend = wstart + poolsize;
    float maxval = -9999;
    int indice = 0;
    ori_data += poolz * height * width;
    for (int h = hstart; h < hend; h++) {
      for (int w = wstart; w < wend; w++) {
         if( ori_data[h * width + w] > maxval){
             maxval = ori_data[h * width + w];
             indice = (h - hstart) * poolsize + w - wstart ;
         }
      }
    }
    pool_data[poolx + pooly * pooled_width + poolz * pooled_height * pooled_width] = maxval;
    indice_data[poolx + pooly * pooled_width + poolz * pooled_height * pooled_width] = indice;
  } 
}


//---------------------------------------------------------------------------------------


__global__ void FMaxPoolBackward(float* reverse_data, float* pool_data, int* indice_data,
    const int num, const int channels, 
    const int height, const int width, 
    const int pooled_height, const int pooled_width,
    const int poolsize, const int poolstrike) {

  int poolx = threadIdx.x + blockIdx.x * blockDim.x;
  int pooly = threadIdx.y + blockIdx.y * blockDim.y;
  int poolz = blockIdx.z;

  if (pooly < pooled_height && poolx < pooled_width) {

    float maxdata = pool_data[poolx + pooly * pooled_width + poolz * pooled_height * pooled_width] ;
    int posit = indice_data[poolx + pooly * pooled_width + poolz * pooled_height * pooled_width] ;

    int hstart = pooly * poolstrike;
    int wstart = poolx * poolstrike;

    int woffset = posit % poolsize;
    int hoffset = int(posit / poolsize);

    int h = hstart + hoffset;
    int w = wstart + woffset;
    reverse_data += poolz * height * width;
    reverse_data[h * width + w] = maxdata;

  }  
}

//--------------------------------------------------------------------------------------------

__global__ void FMaxPoolForwardFix(const float* ori_data, float* pool_data, int* indice_data,
    const int num, const int channels, 
    const int height, const int width, 
    const int pooled_height, const int pooled_width,
    const int poolsize, const int poolstrike) {

  int poolx = threadIdx.x + blockIdx.x * blockDim.x;
  int pooly = threadIdx.y + blockIdx.y * blockDim.y;
  int poolz = blockIdx.z;

  if (pooly < pooled_height && poolx < pooled_width) {

    int posit = indice_data[poolx + pooly * pooled_width + poolz * pooled_height * pooled_width] ;

    int hstart = pooly * poolstrike;
    int wstart = poolx * poolstrike;

    int woffset = posit % poolsize;
    int hoffset = int(posit / poolsize);

    int h = hstart + hoffset;
    int w = wstart + woffset;
    ori_data += poolz * height * width;
    pool_data[poolx + pooly * pooled_width + poolz * pooled_height * pooled_width] = ori_data[h * width + w];

  }  
}
