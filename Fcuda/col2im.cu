// Copyright 2013 

#include <cmath>
#include <cstdlib>
#include <cstring>
#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void im2col(const int n, const float * data_im,
  const int height, const int width, const int imagenum,
  float * data_col) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    int image_in = index/width;
    index -=image_in * width;
    data_col += height * width * image_in +  index * height ;
    data_im += height * image_in + imagenum * height * index;
    for (int i = 0; i < height; ++i) {

        *data_col = data_im[i];
        data_col += 1;
    }
  }
}



