

#include <cmath>
#include <cstdlib>
#include <cstring>
#include "hip/hip_runtime.h"


#include <stdio.h>

// -------------------------------------------------------------------------------------

__global__ void LRNFillScale(const int nthreads, const float* in,
    const int num, const int channels, const int height,
    const int width, const int size, const float alpha_over_size,
    float* scale) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int n = index / width / height;
    int offset = (n * channels * height + h) * width + w;
    int step = height * width;
    in += offset;
    scale += offset;
    int head = 0;
    int pre_pad = (size - 1) / 2;
    int post_pad = size - pre_pad - 1;
    float accum_scale = 0;
    // fill the scale at [n, :, h, w]
    // accumulate values
    while (head < post_pad) {
      accum_scale += in[head * step] * in[head * step];
      ++head;
    }
    // until we reach size, nothing needs to be subtracted
    while (head < size) {
      accum_scale += in[head * step] * in[head * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_scale += in[head * step] * in[head * step];
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      accum_scale -= in[(head - size) * step] * in[(head - size) * step];
      scale[(head - post_pad) * step] = 1. + accum_scale * alpha_over_size;
      ++head;
    }
  }
}

//------------------------------------------------------------------------------

__global__ void LRNComputeOutput(const int nthreads, const float* in,
    const float* scale, const float negative_beta, float* out) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    out[index] = in[index] * pow(scale[index], negative_beta);
  }
}




//-------------------------------------------------------------------------------------

__global__ void LRNComputeDiff(const int nthreads, const float* bottom_data,
    const float* top_data, const float* scale, const float* top_diff,
    const int num, const int channels, const int height,
    const int width, const int size, const float negative_beta,
    const float cache_ratio,
    float* bottom_diff) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int n = index / width / height;
    int offset = (n * channels * height + h) * width + w;
    int step = height * width;
    bottom_data += offset;
    top_data += offset;
    scale += offset;
    top_diff += offset;
    bottom_diff += offset;
    int head = 0;
    int pre_pad = size - (size + 1) / 2;
    int post_pad = size - pre_pad - 1;
    float accum_ratio = 0;
    // accumulate values
    while (head < post_pad) {
      accum_ratio += top_diff[head * step] * top_data[head * step] /
          scale[head * step];
      ++head;
    }
    // until we reach size, nothing needs to be subtracted
    while (head < size) {
      accum_ratio += top_diff[head * step] * top_data[head * step] /
          scale[head * step];
      bottom_diff[(head - post_pad) * step] = top_diff[(head - post_pad) * step]
          * pow(scale[(head - post_pad) * step], negative_beta) - cache_ratio *
          bottom_data[(head - post_pad) * step] * accum_ratio;
      ++head;
    }
    // both add and subtract
    while (head < channels) {
      accum_ratio += top_diff[head * step] * top_data[head * step] /
          scale[head * step];
      accum_ratio -= top_diff[(head - size) * step] *
          top_data[(head - size) * step] / scale[(head - size) * step];
      bottom_diff[(head - post_pad) * step] = top_diff[(head - post_pad) * step]
          * pow(scale[(head - post_pad) * step], negative_beta) - cache_ratio *
          bottom_data[(head - post_pad) * step] * accum_ratio;
      ++head;
    }
    // subtract only
    while (head < channels + post_pad) {
      accum_ratio -= top_diff[(head - size) * step] *
          top_data[(head - size) * step] / scale[(head - size) * step];
      bottom_diff[(head - post_pad) * step] = top_diff[(head - post_pad) * step]
          * pow(scale[(head - post_pad) * step], negative_beta) - cache_ratio *
          bottom_data[(head - post_pad) * step] * accum_ratio;
      ++head;
    }
  }
}


